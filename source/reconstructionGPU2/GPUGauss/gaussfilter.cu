#include "hip/hip_runtime.h"
/*=========================================================================
* GPU accelerated motion compensation for MRI
*
* Copyright (c) 2016 Bernhard Kainz, Amir Alansary, Maria Kuklisova-Murgasova,
* Kevin Keraudren, Markus Steinberger
* (b.kainz@imperial.ac.uk)
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
* FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
* IN THE SOFTWARE.
=========================================================================*/

//! Functionality for performing gaussian filtering

#ifndef GAUSSFILTER_CU
#define GAUSSFILTER_CU

#include <stdio.h>
#include <npp.h>
#include "gaussFilterConvolution.cuh"
//#include "gaussfilter_kernel.cu"
#include "hip/hip_runtime_api.h"


int iDivUp(int a, int b)
{
  return (a + b - 1) / b;
  //return (a % b != 0) ? (a / b + 1) : (a / b);
}


//!/////////////////////////////////////////////////////////////////////////////
//! General Functions
//!/////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
//! Generate 1D Gaussian convolution kernel
//! @param kernel    resulting kernel (necassary memory will be allocated)
//! @param sigma     sigma
//! @param klength   klength of the kernel
////////////////////////////////////////////////////////////////////////////////
int generateGaussianKernel(float** kernel, float sigma, int klength)
{
  // check for valid filter length
  if ((klength % 2) == 0)
  {
    fprintf(stderr, "Error: Convolution Kernel length even\n");
    return -1;
  }

  // allocate memory for kernel
  *kernel = (float*)malloc(sizeof(float) * klength);

  // sum for normalization
  float sum = 0;

  // compute kernel values
  int mid_point = (int)floor(klength/2.0f);
  for( int i = 0; i < klength; i++)
  {
    // generate value
    (*kernel)[i] = exp(-(float)abs(i-mid_point)*(float)abs(i-mid_point)/(2*sigma*sigma));

    // update sum for normalization
    sum += (*kernel)[i];
  }

  // normalize kernel
  for(int i = 0; i < klength; i++)
    (*kernel)[i] /= sum;

  return 0;
}

texture<float, hipTextureType1D, hipReadModeElementType> gaussKernelTex_;


template<int klength>
__global__ void GaussXKernel(hipSurfaceObject_t in, hipSurfaceObject_t out)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  float v = surf2DLayeredread<float>(in, x*4, y, z, hipBoundaryModeClamp);
  
  if(v != -1)
  {
    v = v * tex1Dfetch(gaussKernelTex_, 0);
 
    #pragma unroll
    for(int i = 1; i < (klength+1)/2; ++i)
      v = v + tex1Dfetch(gaussKernelTex_, i) * (max(0.0f,surf2DLayeredread<float>(in, (x + i)*4, y, z, hipBoundaryModeClamp)) + 
                                                max(0.0f,surf2DLayeredread<float>(in, (x - i)*4, y, z, hipBoundaryModeClamp)) );
  }

  surf2DLayeredwrite(v, out, x*4, y, z, hipBoundaryModeZero);
}


__global__ void GaussXKernelGeneral(int klength, hipSurfaceObject_t in, hipSurfaceObject_t out)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  float v = surf2DLayeredread<float>(in, x*4, y, z, hipBoundaryModeClamp);
  
  if(v != -1)
  {
    v = v * tex1Dfetch(gaussKernelTex_, 0);

    for(int i = 1; i < (klength+1)/2; ++i)
      v = v + tex1Dfetch(gaussKernelTex_, i) * (max(0.0f,surf2DLayeredread<float>(in, (x + i)*4, y, z, hipBoundaryModeClamp)) + 
                                                max(0.0f,surf2DLayeredread<float>(in, (x - i)*4, y, z, hipBoundaryModeClamp)) );
  }
 
   surf2DLayeredwrite(v, out, x*4, y, z, hipBoundaryModeZero);
}

template<int klength>
__global__ void GaussYKernel(hipSurfaceObject_t in, hipSurfaceObject_t out)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  float v = surf2DLayeredread<float>(in, x*4, y, z, hipBoundaryModeClamp);
  
  if(v != -1)
  {
    v = v * tex1Dfetch(gaussKernelTex_, 0);
    #pragma unroll
    for(int i = 1; i < (klength+1)/2; ++i)
      v = v + tex1Dfetch(gaussKernelTex_, i) * (max(0.0f,surf2DLayeredread<float>(in, x*4, y + i, z, hipBoundaryModeClamp)) + 
                                                max(0.0f,surf2DLayeredread<float>(in, x*4, y - i, z, hipBoundaryModeClamp)) );
  }
 
  surf2DLayeredwrite(v, out, x*4, y, z, hipBoundaryModeZero);
}

__global__ void GaussYKernelGeneral(int klength, hipSurfaceObject_t in, hipSurfaceObject_t out)
{ 
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  float v = surf2DLayeredread<float>(in, x*4, y, z, hipBoundaryModeClamp);
  
  if(v != -1)
  {
    v = v * tex1Dfetch(gaussKernelTex_, 0);
  
    for(int i = 1; i < (klength+1)/2; ++i)
      v = v + tex1Dfetch(gaussKernelTex_, i) * (max(0.0f,surf2DLayeredread<float>(in, x*4, y + i, z, hipBoundaryModeClamp)) + 
                                                max(0.0f,surf2DLayeredread<float>(in, x*4, y - i, z, hipBoundaryModeClamp)) );
  }
  surf2DLayeredwrite(v, out, x*4, y, z, hipBoundaryModeZero);
}


////////////////////////////////////////////////////////////////////////////////
//! Performes optimized gaussian filtering of a stack of image (x,y direction
//! while slices are stacked up along z
//! @param input          pointer to input image stack
//! @param output         pointer to output image stack
//! @param temp           pointer to temp image stack
//! @param width          width of the image
//! @param height         height of the image
//! @param slices         num slices
//! @param pitchX/Y       image sizes
//! @param num_ch         number of channels in the image
//! @param sigma          sigma parameter to construct kernel
////////////////////////////////////////////////////////////////////////////////
int FilterGaussStack(hipSurfaceObject_t input, hipSurfaceObject_t output, hipSurfaceObject_t temp, 
                 unsigned int width, unsigned int height, unsigned int slices, float sigma)
{
  int ret = 0;

  //determine filter length
  int klength = max(min((int)(sigma*5),MAX_LENGTH_SK),7);
  klength -= 1-klength%2;

  int dev;
  hipGetDevice(&dev);

  static int lastKLength[128] =      {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
  static float lastsigma[128] =      { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  static float* d_GaussKoeffs[128] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  if(lastKLength[dev] != klength || lastsigma[dev] != sigma)
  {
    if(lastKLength[dev] != klength)
    {
      if(d_GaussKoeffs[dev] != 0)
        hipFree(d_GaussKoeffs[dev]);
      hipMalloc(&d_GaussKoeffs[dev], sizeof(float)*(klength+1)/2);
    }

    // generate kernel
    float* kernel = NULL;
    ret = generateGaussianKernel(&kernel, sigma, klength);
    if (ret)
    {
      fprintf(stderr, "Error in CUDA FilterGaussStack(): Could not generate Kernel\n");
      return ret;
    }

    hipMemcpy(d_GaussKoeffs[dev], kernel + klength/2, (klength+1)/2*sizeof(float), hipMemcpyHostToDevice);

    free(kernel);

    

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipBindTexture(0, gaussKernelTex_, d_GaussKoeffs[dev], hipCreateChannelDesc<float>(),(klength+1)/2*sizeof(float)));

    gaussKernelTex_.addressMode[0] = hipAddressModeClamp;
    gaussKernelTex_.filterMode = hipFilterModePoint;
    gaussKernelTex_.normalized = false;

    lastsigma[dev] = sigma;
    lastKLength[dev] = klength;
  }


  //filter (with optimizations for special cases)
  const int blockSize1 = 32;
  const int blockSize2 = 32;

  dim3 blockx(blockSize1, blockSize2);
  dim3 gridx(iDivUp(width, blockSize1), iDivUp(height, blockSize2), slices);
  dim3 blocky(blockSize2, blockSize1);
  dim3 gridy(iDivUp(width, blockSize2), iDivUp(height, blockSize1), slices);

  switch(klength)
  {
  case 7:
    GaussXKernel<7><<<gridx, blockx>>>(input, temp);
    GaussYKernel<7><<<gridy, blocky>>>(temp, output);
    break;
  case 9:
    GaussXKernel<9><<<gridx, blockx>>>(input, temp);
    GaussYKernel<9><<<gridy, blocky>>>(temp, output);
    break;
  case 11:
    GaussXKernel<11><<<gridx, blockx>>>(input, temp);
    GaussYKernel<11><<<gridy, blocky>>>(temp, output);
    break;
  case 13:
    GaussXKernel<13><<<gridx, blockx>>>(input, temp);
    GaussYKernel<14><<<gridy, blocky>>>(temp, output);
    break;
  case 15:
    GaussXKernel<15><<<gridx, blockx>>>(input, temp);
    GaussYKernel<15><<<gridy, blocky>>>(temp, output);
    break;
  default:
    GaussXKernelGeneral<<<gridx, blockx>>>(klength, input, temp);
    GaussYKernelGeneral<<<gridy, blocky>>>(klength, temp, output);
    break;
  }
  return ret;
}


#endif // GAUSSFILTER_CU
