#include "hip/hip_runtime.h"
/*=========================================================================
* GPU accelerated motion compensation for MRI
*
* Copyright (c) 2016 Bernhard Kainz, Amir Alansary, Maria Kuklisova-Murgasova,
* Kevin Keraudren, Markus Steinberger
* (b.kainz@imperial.ac.uk)
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
* FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
* IN THE SOFTWARE.
=========================================================================*/
#include "volume.cuh"
#include "patchBasedVolume.cuh"
#include "patchBasedLayeredSurface3D.cuh"
#include "reconVolume.cuh"
#include "reconConfig.cuh"
#include "pointSpreadFunction.cuh"
#include <irtkImage.h>
#include <thrust/inner_product.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>

//The globally constant point spread function
extern __constant__ PointSpreadFunction<float> _PSF;

template <typename T>
__global__ void patchBasedPSFReconstructionKernel(PatchBasedVolume<T> inputStack, ReconVolume<T> reconstruction, bool useSpx)
{
  //patch based coordinates
  const uint3 pos = make_uint3(blockIdx.x* blockDim.x + threadIdx.x,
    blockIdx.y* blockDim.y + threadIdx.y,
    blockIdx.z* blockDim.z + threadIdx.z);

  uint3 vSize = inputStack.getXYZPatchGridSize();

  if (/*pos.x >= vSize.x || pos.y >= vSize.y ||*/  pos.z >= vSize.z)
    return;

  //from input data with patch calculation
  //float s = inputStack.getValueFromPatchCoords(pos);

  //from patch buffer
  float s = inputStack.getPatchValue(pos);

  if ((s == -1.0f))
    return;

  ImagePatch2D<T> patch = inputStack.getImagePatch2D(pos.z);
  s = s *patch.scale;

  float3 patchPos = make_float3(pos.x, pos.y, 0);
  float3 patchDim = inputStack.getDim();

  float size_inv = 2.0f * _PSF.m_quality_factor / reconstruction.m_dim.x;
  int xDim = round_((patchDim.x * size_inv));
  int yDim = round_((patchDim.y * size_inv));
  int zDim = round_((patchDim.z * size_inv));

  //truncate if value gets close to epsilon
  int dim = MAX_PSF_SUPPORT;
  int centre = (MAX_PSF_SUPPORT - 1) / 2;

  Matrix4<T> combInvTrans = patch.W2I * (patch.InvTransformation * reconstruction.reconstructedI2W);
  float3 psfxyz;
  float3 _psfxyz = reconstruction.reconstructedW2I*(patch.Transformation*  (patch.I2W * patchPos));
  psfxyz = make_float3(round_(_psfxyz.x), round_(_psfxyz.y), round_(_psfxyz.z));

  float sume = 0;
  for (unsigned int z = 0; z < dim; z++)
  {
    for (unsigned int y = 0; y < dim; y++)
    {
      float oldPSF = FLT_MAX;
      for (unsigned int x = 0; x < dim; x++)
      {
        float3 ofsPos;
        float psfval = _PSF.getPSFParamsPrecomp(ofsPos, psfxyz, make_int3(x - centre, y - centre, z - centre), combInvTrans, patchPos, patchDim);
        if (abs(oldPSF - psfval) < PSF_EPSILON) continue;
        oldPSF = psfval;

        uint3 apos = make_uint3(ofsPos.x, ofsPos.y, ofsPos.z);
        if (useSpx) {
          if (apos.x < reconstruction.m_size.x && apos.y < reconstruction.m_size.y && apos.z < reconstruction.m_size.z && patch.spxMask[pos.x+64*pos.y]=='1') { sume += psfval; }
        }else{
          if (apos.x < reconstruction.m_size.x && apos.y < reconstruction.m_size.y && apos.z < reconstruction.m_size.z) { sume += psfval; }
        }

      }
    }
  }

  // printf("patchDim.x = %f , patchDim.y = %f , patchDim.z = %f,sume %f  \n", patchDim.x, patchDim.y , patchDim.z, sume);

  //fix for crazy values at the border -> too accurate ;)
  if ((sume > PSF_EPSILON) || isnan(sume) )
  {
    inputStack.setPSFsumsValue(pos, sume);
  }
  else
  {
    return;
  }

  for (unsigned int z = 0; z < dim; z++)
  {
    for (unsigned int y = 0; y < dim; y++)
    {
      float oldPSF = FLT_MAX;
      for (unsigned int x = 0; x < dim; x++)
      {
        float3 ofsPos;
        float psfval = _PSF.getPSFParamsPrecomp(ofsPos, psfxyz, make_int3(x - centre, y - centre, z - centre), combInvTrans, patchPos, patchDim);
        if (abs(oldPSF - psfval) < PSF_EPSILON) continue;
        oldPSF = psfval;

        uint3 apos = make_uint3(round_(ofsPos.x), round_(ofsPos.y), round_(ofsPos.z)); //NN
        if (apos.x < reconstruction.m_size.x && apos.y < reconstruction.m_size.y && apos.z < reconstruction.m_size.z
          && reconstruction.m_d_mask[apos.x + apos.y*reconstruction.m_size.x + apos.z*reconstruction.m_size.x*reconstruction.m_size.y] != 0)
        {
          psfval /= sume;
          reconstruction.addReconVolWeightValue(apos, psfval);
          reconstruction.addReconValue(apos, s*psfval);//*psfval
        }

      }
    }
  }

}

template <typename T>
void patchBasedPSFReconstruction_gpu(int cuda_device, PatchBasedVolume<T> & inputStack, ReconVolume<T> & reconstruction, bool useSpx = false)
{
  printf("patchBasedPSFReconstruction_gpu\n");

  //TODO patch batch wise for kernel 2s watchdogs necesary?
  checkCudaErrors(hipSetDevice(cuda_device));

  dim3 blockSize3 = dim3(8, 8, 8); //max 1024 threads
  dim3 gridSize3  = divup(dim3(inputStack.getXYZPatchGridSize().x, inputStack.getXYZPatchGridSize().y,
    inputStack.getXYZPatchGridSize().z), blockSize3);
  patchBasedPSFReconstructionKernel<T> << <gridSize3, blockSize3 >> >(inputStack, reconstruction, useSpx);
  CHECK_ERROR(patchBasedPSFReconstructionKernel);
  checkCudaErrors(hipDeviceSynchronize());
}

template void patchBasedPSFReconstruction_gpu<float>(int cuda_device, PatchBasedVolume<float> & inputStack, ReconVolume<float> & reconstruction, bool useSpx);
template void patchBasedPSFReconstruction_gpu<double>(int cuda_device, PatchBasedVolume<double> & inputStack, ReconVolume<double> & reconstruction, bool useSpx);
